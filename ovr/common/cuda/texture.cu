#include "hip/hip_runtime.h"
//. ======================================================================== //
//.                                                                          //
//. Copyright 2019-2022 Qi Wu                                                //
//.                                                                          //
//. Licensed under the MIT License                                           //
//.                                                                          //
//. ======================================================================== //
#include "cuda_misc.h"
#include "cuda_math.h"
#include "texture.h"

#include <assert.h>
#include <math.h>

////////////////////
// Kernel helpers //
////////////////////

#ifdef __NVCC__
#define UTIL_CUDA_HOST_DEVICE __host__ __device__
#else
#define UTIL_CUDA_HOST_DEVICE
#endif

// A key benefit of using the new surface objects is that we don't need any global
// binding points anymore. We can directly pass them as function arguments.

__global__ void
generate_mipmaps_device(uint32_t imageW, uint32_t imageH, hipSurfaceObject_t mipOutput, hipTextureObject_t mipInput)
{
  uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
  uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

  float px = 1.0 / float(imageW);
  float py = 1.0 / float(imageH);

  if ((x < imageW) && (y < imageH)) {
    // take the average of 4 samples

    // we are using the normalized access to make sure non-power-of-two textures
    // behave well when downsized.
    float4 color = (tex2D<float4>(mipInput, (x + 0) * px, (y + 0) * py)) + (tex2D<float4>(mipInput, (x + 1) * px, (y + 0) * py)) +
                   (tex2D<float4>(mipInput, (x + 1) * px, (y + 1) * py)) + (tex2D<float4>(mipInput, (x + 0) * px, (y + 1) * py));

    color /= 4.0;

    surf2Dwrite(color, mipOutput, x * sizeof(float4), y);
  }
}

void
generate_mipmaps(hipMipmappedArray_t mipmapArray, hipExtent size)
{
  size_t width = size.width;
  size_t height = size.height;

  uint32_t level = 0;

  while (width != 1 || height != 1) {
    width /= 2;
    width = MAX((size_t)1, width);
    height /= 2;
    height = MAX((size_t)1, height);

    hipArray_t levelFrom;
    CUDA_CHECK(hipGetMipmappedArrayLevel(&levelFrom, mipmapArray, level));
    hipArray_t levelTo;
    CUDA_CHECK(hipGetMipmappedArrayLevel(&levelTo, mipmapArray, level + 1));

    hipExtent levelToSize;
    CUDA_CHECK(hipArrayGetInfo(NULL, &levelToSize, NULL, levelTo));
    assert(levelToSize.width == width);
    assert(levelToSize.height == height);
    assert(levelToSize.depth == 0);

    // generate texture object for reading
    hipResourceDesc texRes;
    memset(&texRes, 0, sizeof(hipResourceDesc));
    texRes.resType = hipResourceTypeArray;
    texRes.res.array.array = levelFrom;

    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(hipTextureDesc));
    texDesc.filterMode = hipFilterModeLinear;
    texDesc.normalizedCoords = true;
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.addressMode[2] = hipAddressModeClamp;

    hipTextureObject_t texInput;
    CUDA_CHECK(hipCreateTextureObject(&texInput, &texRes, &texDesc, NULL));

    // generate surface object for writing
    hipResourceDesc surfRes;
    memset(&surfRes, 0, sizeof(hipResourceDesc));
    surfRes.resType = hipResourceTypeArray;
    surfRes.res.array.array = levelTo;

    hipSurfaceObject_t surfOutput;
    CUDA_CHECK(hipCreateSurfaceObject(&surfOutput, &surfRes));

    // run mipmap kernel
    util::bilinear_kernel(generate_mipmaps_device, 0, /*stream=*/0, (uint32_t)width, (uint32_t)height, surfOutput, texInput);

    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDestroySurfaceObject(surfOutput));
    CUDA_CHECK(hipDestroyTextureObject(texInput));

    level++;
  }
}

uint32_t
get_mipmap_levels(hipExtent size)
{
  size_t sz = MAX(MAX(size.width, size.height), size.depth);
  uint32_t levels = 0;
  while (sz) {
    sz /= 2;
    levels++;
  }
  return levels;
}

hipTextureObject_t
create_mipmap_rgba32f_texture(void* data, int width, int height)
{
  // how many mipmaps we need
  hipExtent extent;
  extent.width = width;
  extent.height = height;
  extent.depth = 0;
  uint32_t levels = get_mipmap_levels(extent);

  hipChannelFormatDesc desc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
  hipMipmappedArray_t mipmapArray;
  CUDA_CHECK(hipMallocMipmappedArray(&mipmapArray, &desc, extent, levels));

  // upload level 0
  hipArray_t level0;
  CUDA_CHECK(hipGetMipmappedArrayLevel(&level0, mipmapArray, 0));

  hipMemcpy3DParms copyParams = { 0 };
  copyParams.srcPtr = make_hipPitchedPtr(data, width * 4 * sizeof(float), width, height);
  copyParams.dstArray = level0;
  copyParams.extent = extent;
  copyParams.extent.depth = 1;
  copyParams.kind = hipMemcpyHostToDevice;
  CUDA_CHECK(hipMemcpy3D(&copyParams));

  // compute rest of mipmaps based on level 0
  generate_mipmaps(mipmapArray, extent);

  // generate bindless texture object
  hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(hipResourceDesc));
  resDesc.resType = hipResourceTypeMipmappedArray;
  resDesc.res.mipmap.mipmap = mipmapArray;

  hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(hipTextureDesc));
  texDesc.filterMode = hipFilterModeLinear;
  texDesc.mipmapFilterMode = hipFilterModeLinear;
  texDesc.normalizedCoords = true;
  texDesc.addressMode[0] = hipAddressModeClamp;
  texDesc.addressMode[1] = hipAddressModeClamp;
  texDesc.addressMode[2] = hipAddressModeClamp;
  texDesc.maxMipmapLevelClamp = float(levels - 1);

  hipTextureObject_t texture;
  CUDA_CHECK(hipCreateTextureObject(&texture, &resDesc, &texDesc, NULL));
  return texture;
}

hipTextureObject_t
create_pitch2d_rgba32f_texture(void* data, int width, int height)
{
  // Second step: create a cuda texture out of this image. It'll be used to generate training 
  // data efficiently on the fly
  hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypePitch2D;
  resDesc.res.pitch2D.devPtr = data;
  resDesc.res.pitch2D.desc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
  resDesc.res.pitch2D.width = width;
  resDesc.res.pitch2D.height = height;
  resDesc.res.pitch2D.pitchInBytes = width * 4 * sizeof(float);

  hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.filterMode = hipFilterModeLinear;
  texDesc.normalizedCoords = true;
  texDesc.addressMode[0] = hipAddressModeClamp;
  texDesc.addressMode[1] = hipAddressModeClamp;
  texDesc.addressMode[2] = hipAddressModeClamp;

  hipResourceViewDesc viewDesc;
  memset(&viewDesc, 0, sizeof(viewDesc));
  viewDesc.format = hipResViewFormatFloat4;
  viewDesc.width = width;
  viewDesc.height = height;

  hipTextureObject_t texture;
  CUDA_CHECK(hipCreateTextureObject(&texture, &resDesc, &texDesc, &viewDesc));
  return texture;
}
